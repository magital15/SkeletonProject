#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#define TPB 32

// Commented out because this is currently working in the main.
/*
#pragma region gcd / chinese remainder thm helper methods, reconstruct kernel and launcher


// C function for extended Euclidean Algorithm
// code authors: GeeksForGeeks
__device__
int gcdExtended(int a, int b, int *x, int *y)
{
	// Base Case
	if (a == 0)
	{
		*x = 0, *y = 1;
		return b;
	}

	int x1, y1; // To store results of recursive call
	int gcd = gcdExtended(b%a, a, &x1, &y1);

	// Update x and y using results of recursive
	// call
	*x = y1 - (b / a) * x1;
	*y = x1;

	return gcd;
}

// Function to find modulo inverse of a
// PRECONDITION: a and m are coprime
// code authors: GeeksForGeeks
__device__
int2 modInverse(int a, int m)
{
	int x, y;
	int g = gcdExtended(a, m, &x, &y);
	if (g != 1) {
		printf("Inverse doesn't exist for %d, %d", a, m);
		return int2{ 0, 0 };
	}
	else
	{
		// m is added to handle negative x
		int res = (x%m + m) % m;
		int otherRes = (y%a + a) % a;
		

		//printf("%d * %d = 1 (mod %d)\n", a, res, m);
		//printf("So %d is the multiplicative inverse of %d (mod %d)\n", res, a, m);
		//printf("%d * %d = 1 (mod %d)\n", m, otherRes, a);
		//printf("So %d is the multiplicative inverse of %d (mod %d)\n", otherRes, m, a);

		return int2{ res, otherRes };
	}
}

__device__
int reconstruct(Poly a, int col, int *primeArray) {

	int nextMember = 1;
	int nextPrime = 0;

	int a1 = a.members[nextMember++].coeffs[col];
	int a2 = a.members[nextMember].coeffs[col];

	int p1 = primeArray[nextPrime++];
	int p2 = primeArray[nextPrime];

	int a12 = 0;
	int prevAnswer = -1;

	while (nextPrime < NUMPRIMES && a12 != prevAnswer) {

		// only enter this on iterations beyond the first one
		if (nextPrime != 1) {
			a1 = a12;
			a2 = a.members[nextMember].coeffs[col];
			p1 = p1*p2;
			p2 = primeArray[nextPrime];
		}

		int2 multiplicativeInverses = modInverse(p1, p2);
		int k1modp2 = multiplicativeInverses.x;
		int k2modp1 = multiplicativeInverses.y;

		prevAnswer = a12;
		a12 = (a2*k1modp2*p1 + a1*k2modp1*p2) % (p1*p2);

		nextMember++;
		nextPrime++;
	}

	return a12;
}

void reconstructKernel(Poly a, int *primes, int size)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i >= size) return;
	a.members[0].coeffs[i] = reconstruct(a, i, primes); // POINTER ERROR
}

void reconstructPoly(Poly in, int* primes)
{
	int len = in.length;

	// Declare pointers to device arrays
	int *d_in = 0;
	int *d_primes = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_in, len*sizeof(int));
	hipMalloc(&d_primes, NUMPRIMES*sizeof(int));

	hipMemcpy(d_primes, primes, NUMPRIMES*sizeof(int),hipMemcpyHostToDevice);
	
	// HOW TO ACTUALLY MOVE ALL DATA OVER?

	// Do this for all polys in Polyset
	for (int i = 1; i < NUMPRIMES + 1; i++) {
		// Copy input data from host to device
		hipMemcpy(d_in, in.members[i].coeffs, len*sizeof(int),
			hipMemcpyHostToDevice);
	}

	//paralelize by coefficient number
	for (int i = 0; i < len; i++) {
		// Launch kernel to compute and store modded polynomial values
		reconstructKernel <<<(len + TPB - 1) / TPB, TPB >> >(d_in, primes, NUMPRIMES);
	}
	
	// Copy results from device to host
	hipMemcpy(in.members[0].coeffs, d_in, len*sizeof(int),	hipMemcpyDeviceToHost);
	

	// Free the memory allocated for device arrays
	hipFree(d_in);
	hipFree(primes);
}
#pragma endregion 
*/

__device__
int getRemainder(int coefficient, int mod)
{
	coefficient = coefficient % mod;
	coefficient += mod;
	return coefficient % mod;
}

__device__
int getMult(int a, int b)
{
	return a * b;
}

__global__
void takeMod(int* d_out, int* d_in, int mod)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = getRemainder(x, mod);
}

__global__
void addMods(int* d_out, int* d_a, int* d_b, int mod)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_a[i];
    const float y = d_b[i];
	d_out[i] = getRemainder(x + y, mod);
}

__global__
void scalarMultMods(int* d_out, int* d_in, int scalar, int mod)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = getRemainder(x*scalar, mod);
}

// expects d_out to be large enough to hold i+monomial elements
// not sure if works for negative scalars
__global__
void monomialScalarMultMods(int* d_out, int* d_in, int scalar, int monomial, int mod)
{
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const float x = d_in[i];
        d_out[i + monomial] = getRemainder(x*scalar, mod);
}

__global__
void minusMult(int* d_out, int* d_in, int scalar, int mod)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = getMult(x, scalar);
}

void getMods(Poly in, int* primes)
{
	int len = in.length;

	// Declare pointers to device arrays
	int *d_in = 0;
	int *d_out = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_in, len*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));

	// Do this for all polys in Polyset
	for (int i = 1; i < NUMPRIMES+1; i++) {
		// Copy input data from host to device
		hipMemcpy(d_in, in.members[0].coeffs, len*sizeof(int), 
				   hipMemcpyHostToDevice);
  
		// Launch kernel to compute and store modded polynomial values
		takeMod<<<(len + TPB - 1)/TPB, TPB>>>(d_out, d_in, 
											  primes[i-1]);

		// Copy results from device to host
		hipMemcpy(in.members[i].coeffs, d_out, len*sizeof(int), 
				   hipMemcpyDeviceToHost);
	}
  
	// Free the memory allocated for device arrays
	hipFree(d_in);
	hipFree(d_out);
}

void addPolys(Poly a, Poly b, Poly c, int* primes)
{
	int len;
	if (a.length > b.length)
	{
		len = a.length;
	}
	else
	{
		len = b.length;
	}

	// Declare pointers to device arrays
	int *d_a = 0;
	int *d_b = 0;
	int *d_out = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_a, len*sizeof(int));
	hipMalloc(&d_b, len*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));

	// Do this for all polys in Polyset
	for (int i = 1; i < NUMPRIMES+1; i++) {
		// Copy input data from host to device
		hipMemcpy(d_a, a.members[i].coeffs, len*sizeof(int), 
				   hipMemcpyHostToDevice);
		hipMemcpy(d_b, b.members[i].coeffs, len*sizeof(int), 
				   hipMemcpyHostToDevice);
  
		// Launch kernel to compute and store modded polynomial values
		addMods<<<(len + TPB - 1)/TPB, TPB>>>(d_out, d_a, d_b, 
											  primes[i-1]);

		// reconstruct answer

		// Copy results from device to host
		hipMemcpy(c.members[i].coeffs, d_out, len*sizeof(int), 
				   hipMemcpyDeviceToHost);
	}
  
	// Free the memory allocated for device arrays
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
}

void scalarMultPoly(Poly in, Poly out, int scalar, int* primes)
{
	int len = in.length;

	// Declare pointers to device arrays
	int *d_in = 0;
	int *d_out = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_in, len*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));

	// Do this for all polys in Polyset
	for (int i = 1; i < NUMPRIMES+1; i++) {
		// Copy input data from host to device
		hipMemcpy(d_in, in.members[i].coeffs, len*sizeof(int), 
				   hipMemcpyHostToDevice);
  
		// Launch kernel to compute and store modded polynomial values
		scalarMultMods<<<(len + TPB - 1)/TPB, TPB>>>(d_out, d_in,
											scalar, primes[i-1]);

		// Copy results from device to host
		hipMemcpy(out.members[i].coeffs, d_out, len*sizeof(int), 
				   hipMemcpyDeviceToHost);
	}
  
	// Free the memory allocated for device arrays
	hipFree(d_in);
	hipFree(d_out);
}

void subtractPolys(Poly a, Poly b, Poly c, int* primes)
{
	scalarMultPoly(b, c, -1, primes);
	addPolys(a, c, c, primes);

//	Above is same as Below?

/*
	int len = a.length;
l
	// Declare pointers to device arrays
	int *d_a = 0;
	int *d_b = 0;
	int *d_out = 0;

temp	//Allocate memory for device arrays
	hipMalloc(&d_a, len*sizeof(int));
	hipMalloc(&d_b, len*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));
o
	// Do this for all polys in Polyset
	for (int i = 1; i < NUMPRIMES+1; i++) {
		// Copy input data from host to device
		hipMemcpy(d_a, a.members[i].coeffs, len*sizeof(int), 
				   hipMemcpyHostToDevice);
	hipMemcpy(d_b, b.members[i].coeffs, len*sizeof(int), 
				   hipMemcpyHostToDevice);
  
		// Launch kernel to compute and store modded polynomial values
		minusMult<<<(len + TPB - 1)/TPB, TPB>>>(d_out, d_b, -1, 
											    primes[i-1]);
		addMods<<<(len + TPB - 1)/TPB, TPB>>>(d_out, d_a, d_out, 
											  primes[i-1]);

		// Copy results from device to host
		hipMemcpy(c.members[i].coeffs, d_out, len*sizeof(int), 
				   hipMemcpyDeviceToHost);
	}
  
	// Free the memory allocated for device arrays
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
*/
}

void multiplyPolys(Poly a, Poly b, Poly c, int* primes)
{
	int len = c.length;
	Poly shorterPoly = a.length <= b.length ? a : b;
	Poly longerPoly = a.length <= b.length ? b : a;

	// Declare pointers to device arrays
	int *d_a = 0;
	int *d_b = 0;
	int *d_out = 0;
	int *d_temp = 0;

	//Allocate memory for device arrays

	hipMalloc(&d_a, longerPoly.length*sizeof(int));
	hipMalloc(&d_b, shorterPoly.length*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));
	cudaCalloc(&d_temp, len*sizeof(int));

	// Do this for all polys in Polyset
	for (int i = 0; i < NUMPRIMES; i++) {
		// Copy input data from host to device
		hipMemcpy(d_a, longerPoly.members[i].coeffs, longerPoly.length*sizeof(int), 
				   hipMemcpyHostToDevice);
		hipMemcpy(d_b, shorterPoly.members[i].coeffs, shorterPoly.length*sizeof(int), 
				   hipMemcpyHostToDevice);
  		
		for (int j = 0; j < shorterPoly.length; j++) {
			// Launch kernel to compute and store modded polynomial values
			monomialScalarMultMods<<<(len+ TPB - 1)/TPB, TPB>>>(d_temp, d_a, d_b[j], j, primes[i]);
			addMods<<<(len + TPB - 1)/TPB, TPB>>>(d_out, d_temp, d_out, primes[i]);
		}
		// Copy results from device to host
		hipMemcpy(c.members[i].coeffs, d_out, len*sizeof(int), 
				   hipMemcpyDeviceToHost);
	}
  
	// Free the memory allocated for device arrays
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
        hipFree(d_temp);
}

__device__
int2 LCM(int2 a, int2 b){
	// by default, return the product of the two numbers and the larger of the two powers
	return new int2 {a.x*b.x, a.y >= b.y ? a.y : b.y)};
}

void sPoly(Poly a, Poly b, Poly c, int* primes)
{
	int len = c.length; // should be 1 less than longerPoly.length
	
	// Declare pointers to device arrays
	int *d_a = 0;
	int *d_b = 0;
	int *d_out = 0;
	int *d_tempA = 0;
	int *d_tempB = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_a, a.length*sizeof(int));
	hipMalloc(&d_b, b.length*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));
	cudaCalloc(&d_tempA, len*sizeof(int));
	cudaCalloc(&d_tempB, len*sizeof(int));

	// Do this for all polys in Polyset
	for (int i = 0; i < NUMPRIMES; i++) {
		// Copy input data from host to device
		hipMemcpy(d_a, a.members[i].coeffs, a.length*sizeof(int), 
				   hipMemcpyHostToDevice);
		hipMemcpy(d_b, b.members[i].coeffs, b.length*sizeof(int), 
				   hipMemcpyHostToDevice);
  		
		// find LCM of highest power monomial in a and b.
		int2 lastA = {a.members[i].coeffs[a.length-1], a.length-1};
		int2 lastB = {b.members[i].coeffs[b.length-1], b.length-1};
		int LCM = LCM(lastA, lastB);
		
		int aScalar = LCM.x / lastA.x;
		int aMonomial = LCM.y - lastA.y;
		int bScalar = LCM.x / lastB.x;
		int bMonomial = LCM.y = lastB.y;
		
		// multiply both a and b by the requisite scale factor to get the last term to equal the LSM
		monomialScalarMultMods<<<(len + TPB - 1)/TPB, TPB>>>(d_tempA, d_a, aScalar, aMonomial, primes[i]);
		monomialScalarMultMods<<<(len + TPB - 1)/TPB, TPB>>>(d_tempB, d_b, bScalar, bMonomial, primes[i]);
				
		// return a-b 
		// does this work, allowing negative mods??
		scalarMultMods<<<(len + TPB - 1)/TPB, TPB>>>(d_tempB, d_tempB, -1, primes[i]); 
		addMods<<<(len + TPB - 1)/TPB, TPB>>>(d_tempA, d_tempB, d_out, primes[i]);

		// Copy results from device to host
		hipMemcpy(c.members[i].coeffs, d_out, len*sizeof(int), 
				   hipMemcpyDeviceToHost);
	}
  
	// Free the memory allocated for device arrays
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
        hipFree(d_tempA);
	hipFree(d_tempB);
}
