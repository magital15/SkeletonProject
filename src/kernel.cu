#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#define TPB 32

// DEVICE WORK FUNCTIONS
__device__
int getRemainder(int coefficient, int mod)
{
	coefficient = coefficient % mod;
	coefficient += mod;
	return coefficient % mod;
}

__global__
void takeMod(int* d_out, int* d_in, int mod)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = getRemainder(x, mod);
}

__global__
void addMods(int* d_out, int* d_a, int* d_b, int mod, int size)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i > size) return;
	const float x = d_a[i];
    const float y = d_b[i];
	d_out[i] = getRemainder(x + y, mod);
}

__global__
void scalarMultMods(int* d_out, int* d_in, int scalar, int mod, int size)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i >= size) return;
	const float x = d_in[i];
	d_out[i] = getRemainder(x*scalar, mod);
}

// expects d_out to be large enough to hold i+monomial elements
// not sure if works for negative scalars
__global__
void monomialScalarMultMods(int* d_out, int* d_in, int scalar, int monomial, int mod, int size)
{
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i >= size) return;
        const float x = d_in[i];
        d_out[i + monomial] = getRemainder(x*scalar, mod);
}

// KERNEL WRAPPER FUNCTIONS
void getMods(Poly in, int* primes) {
	int len = in.length;

	// Declare pointers to device arrays
	int *d_in = 0;
	int *d_out = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_in, len*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));

	// Take Mod of each member
	for (int i = 1; i <= NUMPRIMES; i++) {
		// Copy input data from host to device
		hipMemcpy(d_in, in.members[0].coeffs, len*sizeof(int), hipMemcpyHostToDevice);

		// Launch kernel to compute and store modded polynomial values
		takeMod<<<(len + TPB - 1)/TPB, TPB>>>(d_out, d_in, primes[i-1]);

		// Copy results from device to host
		hipMemcpy(in.members[i].coeffs, d_out, len*sizeof(int), hipMemcpyDeviceToHost);
	}
	// Free the memory allocated for device arrays
	hipFree(d_in);
	hipFree(d_out);
}

Poly addPolys(Poly a, Poly b, int* primes) {
	// Deal with lengths
	int len = a.length >= b.length ? a.length : b.length;
	a = a.length < b.length ? copyIntoBigger(a, b.length) : a;
	b = a.length > b.length ? copyIntoBigger(b, a.length) : b;
	Poly result = makePolyGivenLength(len);

	// Declare pointers to device arrays
	int *d_a = 0;
	int *d_b = 0;
	int *d_out = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_a, len*sizeof(int));
	hipMalloc(&d_b, len*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));

	// Do this for all members
	for (int i = 1; i <= NUMPRIMES; i++) {
		// Copy input data from host to device
		hipMemcpy(d_a, a.members[i].coeffs, len*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_b, b.members[i].coeffs, len*sizeof(int), hipMemcpyHostToDevice);

		// Launch kernel to compute and store modded polynomial values
		addMods<<<(len + TPB - 1)/TPB, TPB>>>(d_out, d_a, d_b, primes[i-1], len);

		// Copy results from device to host
		hipMemcpy(result.members[i].coeffs, d_out, len*sizeof(int), hipMemcpyDeviceToHost);
	}
	// Free the memory allocated for device arrays
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);

	return result;
}

Poly scalarMultPoly(Poly in, int scalar, int* primes) {
	int len = in.length;
	Poly result = makePolyGivenLength(len);

	// Declare pointers to device arrays
	int *d_in = 0;
	int *d_out = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_in, len*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));

	// Do this for all members
	for (int i = 1; i <= NUMPRIMES; i++) {
		// Copy input data from host to device
		hipMemcpy(d_in, in.members[i].coeffs, len*sizeof(int), hipMemcpyHostToDevice);

		// Launch kernel to compute and store modded polynomial values
		scalarMultMods<<<(len + TPB - 1)/TPB, TPB>>>(d_out, d_in, scalar, primes[i-1], len);

		// Copy results from device to host
		hipMemcpy(result.members[i].coeffs, d_out, len*sizeof(int), hipMemcpyDeviceToHost);
	}
	// Free the memory allocated for device arrays
	hipFree(d_in);
	hipFree(d_out);

	return result;
}

// Can be accelerated by not copying in between two arithmetic operators
Poly subtractPolys(Poly a, Poly b, int* primes) {
	Poly result = scalarMultPoly(b, -1, primes);
	result = addPolys(a, result, primes);
	return result;
}

Poly multiplyPolys(Poly a, Poly b, int* primes) {
	// Deal with lengths
	int len = a.length + b.length - 1;
	Poly result = makePolyGivenLength(len);
	Poly shorterPoly = a.length <= b.length ? a : b;
	Poly longerPoly = a.length <= b.length ? b : a;
	shorterPoly = copyIntoBigger(shorterPoly, len);
	longerPoly = copyIntoBigger(longerPoly, len);
	
	// Declare pointers to device arrays
	int *d_a = 0;
	int *d_b = 0;
	int *d_out = 0;
	int *d_temp = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_a, longerPoly.length*sizeof(int));
	hipMalloc(&d_b, shorterPoly.length*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));
	hipMalloc(&d_temp, len*sizeof(int));

	// Do this for all members
	for (int i = 1; i <= NUMPRIMES; i++) {
		// Copy input data from host to device
		hipMemcpy(d_a, longerPoly.members[i].coeffs, longerPoly.length*sizeof(int), hipMemcpyHostToDevice);
  		hipMemset(d_out, 0, len*sizeof(int));

		for (int j = 0; j < shorterPoly.length; j++) {
			hipMemset(d_temp, 0, len*sizeof(int));
			// Launch kernel to compute and store modded polynomial values
			monomialScalarMultMods<<<(len + TPB - 1) / TPB, TPB>>>(d_temp, d_a, shorterPoly.members[i].coeffs[j], j, primes[i-1], longerPoly.length);
			addMods<<<(len + TPB - 1) / TPB, TPB>>>(d_out, d_temp, d_out, primes[i-1], len);
		}
		// Copy results from device to host
		hipMemcpy(result.members[i].coeffs, d_out, len*sizeof(int), hipMemcpyDeviceToHost);
	}
	// Free the memory allocated for device arrays
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
    hipFree(d_temp);
	
	return result;
}

int2 LCM(int2 a, int2 b) {
	// Return the product of the two numbers and the larger of the two powers
	int2 c = {a.x*b.x, a.y >= b.y ? a.y : b.y};
	return c;
}

Poly sPoly(Poly a, Poly b, int* primes) {
	int len = a.length >= b.length ? a.length-1 : b.length-1;
	Poly result = makePolyGivenLength(len);

	// Declare pointers to device arrays
	int *d_a = 0;
	int *d_b = 0;
	int *d_out = 0;
	int *d_tempA = 0;
	int *d_tempB = 0;

	//Allocate memory for device arrays
	hipMalloc(&d_a, (len+1)*sizeof(int));
	hipMalloc(&d_b, (len+1)*sizeof(int));
	hipMalloc(&d_out, len*sizeof(int));
	hipMalloc(&d_tempA, (len+1)*sizeof(int));
	hipMalloc(&d_tempB, (len+1)*sizeof(int));
	
	// Do this for all members
	for (int i = 1; i <= NUMPRIMES; i++) {
		hipMemset(d_a, 0, (len+1)*sizeof(int));
		hipMemset(d_b, 0, (len+1)*sizeof(int));
		hipMemset(d_out, 0, len*sizeof(int));
		hipMemset(d_tempA, 0, (len+1)*sizeof(int));
		hipMemset(d_tempB, 0, (len+1)*sizeof(int));

		// Copy input data from host to device
		hipMemcpy(d_a, a.members[i].coeffs, a.length*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_b, b.members[i].coeffs, b.length*sizeof(int), hipMemcpyHostToDevice);
  		
		// find LCM of highest power monomial in a and b.
		int2 lastA = {a.members[i].coeffs[a.length-1], a.length-1};
		int2 lastB = {b.members[i].coeffs[b.length-1], b.length-1};
		int2 lcm = LCM(lastA, lastB);

		int aScalar = lcm.x / lastA.x;
		int aMonomial = lcm.y - lastA.y;
		int bScalar = lcm.x / lastB.x;
		int bMonomial = lcm.y - lastB.y;
		int currPrime = primes[i-1];
		// Multiply a and b by the requisite scale factor to get the last term to equal the LSM
		monomialScalarMultMods<<<(a.length + TPB - 1)/TPB, TPB>>>(d_tempA, d_a, aScalar, aMonomial, currPrime, len+1);
		monomialScalarMultMods<<<(b.length + TPB - 1) / TPB, TPB>>>(d_tempB, d_b, bScalar, bMonomial, currPrime, len+1);
						
		// Return a - b 
		scalarMultMods<<<(len + TPB - 1) / TPB, TPB >>>(d_tempB, d_tempB, -1, currPrime, len);
		addMods<<<(len + TPB - 1) / TPB, TPB>>>(d_out, d_tempB, d_tempA, currPrime, len);

		// Copy results from device to host
		hipMemcpy(result.members[i].coeffs, d_out, len*sizeof(int), hipMemcpyDeviceToHost);
	}
  	// Free the memory allocated for device arrays
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
    hipFree(d_tempA);
	hipFree(d_tempB);
	
	return result;
}

Poly exponentiate(Poly a, int exp, int* primeArray) {
	Poly result = copyIntoBigger(a, a.length);
	for(int i = 1; i < exp; i++) {
		result = multiplyPolys(a, result, primeArray);
	}
	return result;
}

Poly exponentiateGPU(Poly a, int exp, int* primes) {
	int len = a.length + (exp-1)*(a.length-1);
	Poly result = makePolyGivenLength(len);
	
    // Declare pointers to device arrays
    int *d_a = 0;
    int *d_out = 0;
    int *d_temp = 0;

	//Allocate memory for device arrays
    hipMalloc(&d_a, len*sizeof(int));
    hipMalloc(&d_out, len*sizeof(int));
    hipMalloc(&d_temp, len*sizeof(int));

	// Do this for all members
	for (int i = 1; i <= NUMPRIMES; i++) {
		// Reset memory and set lengths
		hipMemset(d_out, 0, len*sizeof(int));
		hipMemset(d_a, 0, len*sizeof(int));
		hipMemcpy(d_a, a.members[i].coeffs, a.length*sizeof(int), hipMemcpyHostToDevice);	
		int currentLen = a.length*2 - 1;
		int otherLen = currentLen - a.length + 1;

		for (int numExp = 1; numExp < exp; numExp++) {
			for (int j = 0; j < a.length; j++) {
				// Launch kernel to compute and store modded polynomial values			
				hipMemset(d_temp, 0, currentLen*sizeof(int));
				monomialScalarMultMods<<<(len + TPB - 1) / TPB, TPB>>>(d_temp, d_a, a.members[i].coeffs[j], j, primes[i-1], otherLen);
				addMods <<<(len + TPB - 1) / TPB, TPB >>>(d_out, d_temp, d_out, primes[i-1], currentLen);
			}
			// Copy d_out into d_a, then reset d_out
			hipMemcpy(d_a, d_out, len*sizeof(int), hipMemcpyDeviceToDevice);
			if (numExp != exp-1) {
				hipMemset(d_out, 0, len*sizeof(int));
			}
			// Update length variables
			otherLen = currentLen;
			currentLen += a.length - 1;
		}
		// Copy results from device to host
		hipMemcpy(result.members[i].coeffs, d_out, len*sizeof(int), hipMemcpyDeviceToHost);
	}
	// Free the memory allocated for device arrays
	hipFree(d_a);
	hipFree(d_out);
    hipFree(d_temp);
	
	return result;
}
