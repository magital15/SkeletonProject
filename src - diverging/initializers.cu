#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <stdlib.h>
#define SAFETYLENGTH

// Function that sets primes to reduce how much code is needed
int* setPrimes(int primes[])
{
	int* result = (int*)calloc(NUMPRIMES, sizeof(int));
	for (int i = 0; i < NUMPRIMES; i++)
	{
		result[i] = primes[i];
	}
	return result;
}

// This function makes original polynomials that will be used for arithmetic
Poly makeOriginalPoly(int coeffArray[], int len, int primes[])
{
	// Create a Poly to store data into
	Poly result = makePolyGivenLength(len);;

	// Copy the coeffArray[] into the new Poly in members[0]
	for (int i = 0; i < result.length; i++)
	{
		result.members[0].coeffs[i] = coeffArray[i];
	}

	// Get the modular results to fill the rest of the members
	getMods(result, primes);

	return result;
}

Poly makePolyGivenLength(int length) {
	// Create a Poly to store data into
	Poly result;

	result.length = length;

	// Allocate memory based on the length found
	for (int i = 0; i < NUMPRIMES + 1; i++)
	{
		result.members[i].coeffs = (int*)calloc(result.length, sizeof(int));
	}

	return result;
}

Poly makeScalarPoly(Poly a)
{
	int length = a.length;
	return makePolyGivenLength(length);
}

// This function initializes a polynomial that will be added or subtracted
// initializes the memory for a poly of length max(length of a, length of b)
Poly makeAddPoly(Poly a, Poly b)
{
	int length = a.length >= b.length ? a.length : b.length;
	return makePolyGivenLength(length);
}

// initializes the memory for a poly of length one less than the product of lengths of a, b
Poly makeMultiplyPoly(Poly a, Poly b)
{
	int length = a.length + b.length - 1;
	return makePolyGivenLength(length);
}

// initializes the memory for a poly with length one less than the max length of a, b
Poly makeSPoly(Poly a, Poly b)
{
	int length = a.length >= b.length ? a.length - 1 : b.length - 1;
	return makePolyGivenLength(length);
}

Poly copyIntoBigger(Poly a, int len)
{
	// Create a Poly to store data into
	Poly result;

	// Give the result the required length
	result.length  = len;

	// Copy the original data into the new poly
	for (int i = 0; i < NUMPRIMES + 1; i++)
	{
		result.members[i].coeffs = (int*)calloc(result.length, sizeof(int));
		for (int j = 0; j < a.length; j++)
		{
			result.members[i].coeffs[j] = a.members[i].coeffs[j];
		}
	}

	return result;
}


Poly makeNewPoly()
{
	Poly result;
	result.length = 1;
	for (int i = 0; i < NUMPRIMES + 1; i++)
	{
		result.members[i].coeffs = (int*)calloc(1, sizeof(int));
		result.members[i].coeffs[0] = 0;
	}
	return result;
}

void printForReconstruction(Poly g, int* primeArray) {
	printf("mp = {");
	for (int j = 1; j <= NUMPRIMES; j++)
	{
		printf("{%d, ", primeArray[j - 1]);
		for (int i = 0; i < g.length - 1; i++)
		{
			printf("%i,", g.members[j].coeffs[i]);
		}
		printf("%i}", g.members[j].coeffs[g.length - 1]);
		if (j != NUMPRIMES)
			printf(",");
		else
			printf("};");
		printf("\n");
	}
}
